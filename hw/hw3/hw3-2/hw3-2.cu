
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#define MY_INF 1073741823
#define BLOCK_SIZE 8

int V, E;
int matrix_size;
int *adjacency_matrix;
size_t result;
__constant__ int d_matrix_size;

int ceil(int a, int b) { return (a + b - 1) / b; }

void input(char* infile) {
  FILE *input_file = fopen(infile, "rb");
  result = fread(&V, sizeof(int), 1, input_file);
  result = fread(&E, sizeof(int), 1, input_file);
  
  matrix_size = ((V / BLOCK_SIZE) + 1 ) * BLOCK_SIZE; // matrix size must be multiple of BLOCK_SIZE

  printf("%d, %d, %d\n", V, E, matrix_size); // mygodimatomato : for checking
  adjacency_matrix = (int *)malloc(matrix_size * matrix_size * sizeof(int)); // set the matrix to 1D array for easier data transfer

  hipHostMalloc((void **)&adjacency_matrix, matrix_size * matrix_size * sizeof(int), hipHostMallocDefault); // pinned memory for faster data transfer
 
  for(int i = 0; i < matrix_size; i++) {
    for(int j = 0; j < matrix_size; j++) {
      if (i == j)
        adjacency_matrix[i * matrix_size + j] = 0;
      else 
        adjacency_matrix[i * matrix_size + j] = MY_INF;
    }
  }

  int edge[3];
  for(int i = 0; i < E; i++) {
    result = fread(edge, sizeof(int), 3, input_file);
    adjacency_matrix[edge[0] * matrix_size + edge[1]] = edge[2];
  }
  fclose(input_file);
}

void output(char* outFileName){
  FILE* outfile = fopen(outFileName, "w");
  for (int i = 0; i < V; i++) {
    for (int j = 0; j < V; j++) {
      if (adjacency_matrix[i * matrix_size + j] > MY_INF)
        adjacency_matrix[i*matrix_size + j] = MY_INF;
      if (i != 0)
        adjacency_matrix[i * V + j] = adjacency_matrix[i * matrix_size + j];
    }
  }

  fwrite(adjacency_matrix, sizeof(int), V * V, outfile);
  fclose(outfile);
}

__global__ void phase1(int* d_dist, int r, int* d_check){
  // Get index
  int j = threadIdx.x;
  int i = threadIdx.y;
  
  // Copy data from global memory to shared memory
  extern __shared__ int shared_memory[];
  shared_memory[i * BLOCK_SIZE + j] = d_dist[(i+r*BLOCK_SIZE) * d_matrix_size + (j+r*BLOCK_SIZE)];
  // d_check[i * BLOCK_SIZE + j] = d_dist[(i+r*BLOCK_SIZE) * d_matrix_size + (j+r*BLOCK_SIZE)]; // mygodimatomato: for checking
  // d_check[i * BLOCK_SIZE + j] = 0; // mygodimatomato: for checking
  __syncthreads();


  // D(i,j) = min(D(i,j), D(i,k)+D(k,j))
  #pragma unroll 8 // mygodimatomato: should changed by BLOCK_SIZE
  for(int k = 0; k < BLOCK_SIZE; k++){
    int i_2_k = shared_memory[i * BLOCK_SIZE + k];
    int k_2_j = shared_memory[k * BLOCK_SIZE + j];

    if (i_2_k + k_2_j < shared_memory[i * BLOCK_SIZE + j])
      shared_memory[i * BLOCK_SIZE + j] = i_2_k + k_2_j;
  }
  // d_check[i * BLOCK_SIZE + j] = shared_memory[i * BLOCK_SIZE + j];

  // writing data back to global memory
  d_dist[(i+r*BLOCK_SIZE) * d_matrix_size + (j+r*BLOCK_SIZE)] = shared_memory[i * BLOCK_SIZE + j];
}

__global__ void phase2(int* d_dist, int r){
  int j = threadIdx.x;
  int i = threadIdx.y;

}

__global__ void phase3(int* d_dist, int r){

}


void block_FW(int* d_dist, int* d_check) {
  int round = ceil(V, BLOCK_SIZE);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  round = 1; // mygodimatomato: for checking
  for (int r = 0; r < round; r++) {
    phase1<<<1, dimBlock, BLOCK_SIZE * BLOCK_SIZE * sizeof(int)>>>(d_dist, r, d_check);
    // phase2<<<>>>();
    // phase3<<<>>>();
  }
}


int main(int argc, char* argv[]) {
  // Read input from input file
  input(argv[1]);

  // Allocate the memory for the matrix in GPU
  int *d_dist;
  hipMalloc((void**)&d_dist, sizeof(int) * matrix_size * matrix_size);
  hipMemcpy(d_dist, adjacency_matrix, sizeof(int) * matrix_size * matrix_size, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_matrix_size), &matrix_size, sizeof(int));
  
  // mygodimatomato: for checking
  int *h_check;
  int *d_check;
  h_check = (int*)malloc(sizeof(int)*BLOCK_SIZE *BLOCK_SIZE);
  hipMalloc((void**)&d_check, sizeof(int)*BLOCK_SIZE *BLOCK_SIZE);

  // mygodimatomato : for checking
  for (int i = 0; i < V; i++) {
    for (int j = 0; j < V; j++){
      if(adjacency_matrix[i * matrix_size + j] == MY_INF)
        printf("INF ");
      else 
        printf("%3d ", adjacency_matrix[i * matrix_size + j]);
    } printf("\n");
  } printf("\n");
  
  // Start executing the block Floyed-Warshall
  block_FW(d_dist, d_check);

  // Copy the outcome back to the adjacency_matrix 
  hipMemcpy(adjacency_matrix, d_dist, sizeof(int) * matrix_size * matrix_size, hipMemcpyDeviceToHost);
  hipMemcpy(h_check, d_check, sizeof(int)*BLOCK_SIZE*BLOCK_SIZE, hipMemcpyDeviceToHost);
  
  output(argv[2]);

  // mygodimatomato : for checking
  int k = 0;
  for (int i = 0; i < V; i++) {
    for (int j = 0; j < V; j++){
      if(adjacency_matrix[k] == MY_INF)
        printf("INF ");
      else
        printf("%3d ", adjacency_matrix[k]);
      k++;
    } printf("\n");
  } printf("\n");

  for(int i = 0; i <BLOCK_SIZE;i++){
    for(int j = 0; j <BLOCK_SIZE; j++){
      if(h_check[i*BLOCK_SIZE+j] == MY_INF)
        printf("INF ");
      else
        printf("%3d ", h_check[i*BLOCK_SIZE+j]);
    }printf("\n");
  }printf("\n");

  // Write output to output file
  return 0;
}