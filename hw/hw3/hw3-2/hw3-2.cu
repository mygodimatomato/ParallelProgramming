
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#define MY_INF 1073741823
#define BLOCK_SIZE 8

int V, E;
int matrix_size;
int *adjacency_matrix;
size_t result;
__constant__ int d_matrix_size;

int ceil(int a, int b) { return (a + b - 1) / b; }

void input(char* infile) {
  FILE *input_file = fopen(infile, "rb");
  result = fread(&V, sizeof(int), 1, input_file);
  result = fread(&E, sizeof(int), 1, input_file);
  
  matrix_size = ((V / BLOCK_SIZE) + 1 ) * BLOCK_SIZE; // matrix size must be multiple of BLOCK_SIZE

  adjacency_matrix = (int *)malloc(matrix_size * matrix_size * sizeof(int)); // set the matrix to 1D array for easier data transfer

  hipHostMalloc((void **)&adjacency_matrix, matrix_size * matrix_size * sizeof(int), hipHostMallocDefault); // pinned memory for faster data transfer
 
  for(int i = 0; i < matrix_size; i++) {
    for(int j = 0; j < matrix_size; j++) {
      if (i == j)
        adjacency_matrix[i * matrix_size + j] = 0;
      else 
        adjacency_matrix[i * matrix_size + j] = MY_INF;
    }
  }

  int edge[3];
  for(int i = 0; i < E; i++) {
    result = fread(edge, sizeof(int), 3, input_file);
    adjacency_matrix[edge[0] * matrix_size + edge[1]] = edge[2];
  }
  fclose(input_file);
}

void output(char* outFileName){
  FILE* outfile = fopen(outFileName, "w");
  for (int i = 0; i < V; i++) {
    for (int j = 0; j < V; j++) {
      if (adjacency_matrix[i * matrix_size + j] >= MY_INF)
        adjacency_matrix[i*matrix_size + j] = MY_INF;
      adjacency_matrix[i * V + j] = adjacency_matrix[i * matrix_size + j];
    }
  }

  fwrite(adjacency_matrix, sizeof(int), V * V, outfile);
  fclose(outfile);
}

__global__ void phase1(int* d_dist, int r){
  // Get index
  int j = threadIdx.x;
  int i = threadIdx.y;
  
  // Copy data from global memory to shared memory
  __shared__ int shared_memory[BLOCK_SIZE * BLOCK_SIZE];
  shared_memory[i * BLOCK_SIZE + j] = d_dist[(i+r*BLOCK_SIZE) * d_matrix_size + (j+r*BLOCK_SIZE)];
  __syncthreads();


  // D(i,j) = min(D(i,j), D(i,k)+D(k,j))
  #pragma unroll 8 // mygodimatomato: should changed by BLOCK_SIZE
  for(int k = 0; k < BLOCK_SIZE; k++){
    int i_2_k = shared_memory[i * BLOCK_SIZE + k];
    int k_2_j = shared_memory[k * BLOCK_SIZE + j];

    if (i_2_k + k_2_j < shared_memory[i * BLOCK_SIZE + j])
      shared_memory[i * BLOCK_SIZE + j] = i_2_k + k_2_j;
  }

  // writing data back to global memory
  d_dist[(i+r*BLOCK_SIZE) * d_matrix_size + (j+r*BLOCK_SIZE)] = shared_memory[i * BLOCK_SIZE + j];
}

__global__ void phase2(int* d_dist, int r){
  int j = threadIdx.x; // col index
  int i = threadIdx.y; // row index
  int i_offset = 0;
  int j_offset = 0;
  int i_2_k, k_2_j;

  // 0 : row block, 1 : col block, 2 : center block
  __shared__ int shared_memory[3 * BLOCK_SIZE * BLOCK_SIZE];

  shared_memory[i * BLOCK_SIZE + j + (BLOCK_SIZE*BLOCK_SIZE)*2] = d_dist[(i+r*BLOCK_SIZE) * d_matrix_size + (j+r*BLOCK_SIZE)];

  if (blockIdx.x == 1) { // col 
    i_offset = BLOCK_SIZE * blockIdx.y; 
    j_offset = BLOCK_SIZE * r;
  } else { // row
    i_offset = BLOCK_SIZE * r;
    j_offset = BLOCK_SIZE * blockIdx.y;
  }

  shared_memory[i * BLOCK_SIZE + j + (BLOCK_SIZE * BLOCK_SIZE) * blockIdx.x] = d_dist[(i+i_offset) * d_matrix_size + j + j_offset];
  __syncthreads();

  #pragma unroll 8 // mygodimatomato: should changed by BLOCK_SIZE
  for (int k = 0; k < BLOCK_SIZE; k++) {
    if (blockIdx.x == 0){
      i_2_k = shared_memory[i * BLOCK_SIZE + k + (BLOCK_SIZE*BLOCK_SIZE)*2];
      k_2_j = shared_memory[k * BLOCK_SIZE + j];
    } else {
      i_2_k = shared_memory[i * BLOCK_SIZE + k + (BLOCK_SIZE * BLOCK_SIZE)];
      k_2_j = shared_memory[k * BLOCK_SIZE + j + (BLOCK_SIZE*BLOCK_SIZE)*2];
    }

    if (shared_memory[i * BLOCK_SIZE + j + (BLOCK_SIZE * BLOCK_SIZE) * blockIdx.x] > i_2_k + k_2_j)
      shared_memory[i * BLOCK_SIZE + j + (BLOCK_SIZE * BLOCK_SIZE) * blockIdx.x] = i_2_k + k_2_j;
  }
    
  d_dist[(i+i_offset) * d_matrix_size + j + j_offset] = shared_memory[i * BLOCK_SIZE + j + (BLOCK_SIZE * BLOCK_SIZE) * blockIdx.x];
}

__global__ void phase3(int* d_dist, int r, int* d_check){
  int j = threadIdx.x; // col index
  int i = threadIdx.y; // row index
  int i_offset = BLOCK_SIZE * blockIdx.x;
  int j_offset = BLOCK_SIZE * blockIdx.y;

  __shared__ int shared_memory[3*BLOCK_SIZE*BLOCK_SIZE];

  // d_dist[(i + i_offset) * d_matrix_size + (j + j_offset)] = blockIdx.y + blockIdx.x;

  shared_memory[i * BLOCK_SIZE + j] = d_dist[(i + i_offset) * d_matrix_size + (j + j_offset)];
  shared_memory[BLOCK_SIZE * (i + BLOCK_SIZE) + j] = d_dist[(i + i_offset) * d_matrix_size + j + r * BLOCK_SIZE];
  shared_memory[BLOCK_SIZE * (i + BLOCK_SIZE*2) + j] = d_dist[(i + r * BLOCK_SIZE) * d_matrix_size + (j + j_offset)];
  __syncthreads();

  #pragma unroll 8
  for(int k = 0; k < BLOCK_SIZE; k++){
    int i_2_k = shared_memory[BLOCK_SIZE * (i + BLOCK_SIZE) + k];
    int k_2_j = shared_memory[BLOCK_SIZE * (k + BLOCK_SIZE*2) + j];
    if (shared_memory[i * BLOCK_SIZE + j] > i_2_k + k_2_j)
      shared_memory[i * BLOCK_SIZE + j] = i_2_k + k_2_j;
  }

  d_dist[(i + i_offset) * d_matrix_size + (j + j_offset)] = shared_memory[i * BLOCK_SIZE + j];
}


void block_FW(int* d_dist, int* d_check) {
  int round = matrix_size/BLOCK_SIZE;
  dim3 num_threads(BLOCK_SIZE, BLOCK_SIZE);
  dim3 phase2_num_blocks(2, round); // one for col, one for row, one block will be redundant, but for the whole performance it doesn't really matters
  dim3 phase3_num_blocks(round, round);

  // round = 1; // mygodimatomato: for checking
  for (int r = 0; r < round; r++) {
    phase1<<<1, num_threads, BLOCK_SIZE * BLOCK_SIZE * sizeof(int)>>>(d_dist, r);
    phase2<<<phase2_num_blocks, num_threads, 3 * BLOCK_SIZE * BLOCK_SIZE * sizeof(int)>>>(d_dist, r);
    phase3<<<phase3_num_blocks, num_threads, 3 * BLOCK_SIZE * BLOCK_SIZE * sizeof(int)>>>(d_dist, r, d_check);
  }
}


int main(int argc, char* argv[]) {
  // Read input from input file
  input(argv[1]);

  // Allocate the memory for the matrix in GPU
  int *d_dist;
  hipMalloc((void**)&d_dist, sizeof(int) * matrix_size * matrix_size);
  hipMemcpy(d_dist, adjacency_matrix, sizeof(int) * matrix_size * matrix_size, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_matrix_size), &matrix_size, sizeof(int));
  
  // mygodimatomato: for checking
  int *h_check;
  int *d_check;
  h_check = (int*)malloc(sizeof(int)*BLOCK_SIZE *BLOCK_SIZE);
  hipMalloc((void**)&d_check, sizeof(int)*BLOCK_SIZE *BLOCK_SIZE);
  
  // Start executing the block Floyed-Warshall
  block_FW(d_dist, d_check);

  // Copy the outcome back to the adjacency_matrix 
  hipMemcpy(adjacency_matrix, d_dist, sizeof(int) * matrix_size * matrix_size, hipMemcpyDeviceToHost);
  // cudaMemcpy(h_check, d_check, sizeof(int)*BLOCK_SIZE*BLOCK_SIZE, cudaMemcpyDeviceToHost);
  
  output(argv[2]);

  // mygodimatomato : for checking
  int k = 0;
  for (int i = 0; i < V; i++) {
    for (int j = 0; j < V; j++){
      if(adjacency_matrix[k] == MY_INF)
        printf(" INF ");
      else
        printf("%4d ", adjacency_matrix[k]);
      k++;
    } printf("\n");
  } printf("\n");

  // for(int i = 0; i <BLOCK_SIZE;i++){
  //   for(int j = 0; j <BLOCK_SIZE; j++){
  //     if(h_check[i*BLOCK_SIZE+j] == MY_INF)
  //       printf(" INF ");
  //     else
  //       printf("%4d ", h_check[i*BLOCK_SIZE+j]);
  //   }printf("\n");
  // }printf("\n");

  // Write output to output file
  return 0;
}