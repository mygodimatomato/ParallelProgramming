
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#define MY_INF 1073741823
#define BLOCK_SIZE 32

int V, E;
int matrix_size;
int *adjacency_matrix;
size_t result;
__constant__ int d_matrix_size;

void input(char* infile) {
  FILE *input_file = fopen(infile, "rb");
  result = fread(&V, sizeof(int), 1, input_file);
  result = fread(&E, sizeof(int), 1, input_file);
  
  matrix_size = ((V / BLOCK_SIZE) + 1 ) * BLOCK_SIZE; // matrix size must be multiple of BLOCK_SIZE

  adjacency_matrix = (int *)malloc(matrix_size * matrix_size * sizeof(int)); // set the matrix to 1D array for easier data transfer

  hipHostMalloc((void **)&adjacency_matrix, matrix_size * matrix_size * sizeof(int), hipHostMallocDefault); // pinned memory for faster data transfer
 
  for(int i = 0; i < matrix_size; i++) {
    for(int j = 0; j < matrix_size; j++) {
      if (i == j)
        adjacency_matrix[i * matrix_size + j] = 0;
      else 
        adjacency_matrix[i * matrix_size + j] = MY_INF;
    }
  }

  int edge[3];
  for(int i = 0; i < E; i++) {
    result = fread(edge, sizeof(int), 3, input_file);
    adjacency_matrix[edge[0] * matrix_size + edge[1]] = edge[2];
  }
  fclose(input_file);
}

void output(char* outFileName){
  FILE* outfile = fopen(outFileName, "w");
  for (int i = 0; i < V; i++) {
    for (int j = 0; j < V; j++) {
      if (adjacency_matrix[i * matrix_size + j] >= MY_INF)
        adjacency_matrix[i*matrix_size + j] = MY_INF;
      adjacency_matrix[i * V + j] = adjacency_matrix[i * matrix_size + j];
    }
  }

  fwrite(adjacency_matrix, sizeof(int), V * V, outfile);
  fclose(outfile);
}

__global__ void phase1(int* d_dist, int r){
  // Get index
  int j = threadIdx.x;
  int i = threadIdx.y*4;
  
  // Copy data from global memory to shared memory
  __shared__ int shared_memory[BLOCK_SIZE * BLOCK_SIZE];
  shared_memory[(i+0) * BLOCK_SIZE + j] = d_dist[(i+0+r*BLOCK_SIZE) * d_matrix_size + (j+r*BLOCK_SIZE)];
  shared_memory[(i+1) * BLOCK_SIZE + j] = d_dist[(i+1+r*BLOCK_SIZE) * d_matrix_size + (j+r*BLOCK_SIZE)];
  shared_memory[(i+2) * BLOCK_SIZE + j] = d_dist[(i+2+r*BLOCK_SIZE) * d_matrix_size + (j+r*BLOCK_SIZE)];
  shared_memory[(i+3) * BLOCK_SIZE + j] = d_dist[(i+3+r*BLOCK_SIZE) * d_matrix_size + (j+r*BLOCK_SIZE)];
  __syncthreads();


  // D(i,j) = min(D(i,j), D(i,k)+D(k,j))
  #pragma unroll  // mygodimatomato: should changed by BLOCK_SIZE
  for(int k = 0; k < BLOCK_SIZE; k++){
    shared_memory[(i+0)*BLOCK_SIZE+j] = min(shared_memory[(i+0)*BLOCK_SIZE+j], shared_memory[(i+0) * BLOCK_SIZE + k] +  shared_memory[k * BLOCK_SIZE + j]);
    shared_memory[(i+1)*BLOCK_SIZE+j] = min(shared_memory[(i+1)*BLOCK_SIZE+j], shared_memory[(i+1) * BLOCK_SIZE + k] + shared_memory[k * BLOCK_SIZE + j]);
    shared_memory[(i+2)*BLOCK_SIZE+j] = min(shared_memory[(i+2)*BLOCK_SIZE+j], shared_memory[(i+2) * BLOCK_SIZE + k] + shared_memory[k * BLOCK_SIZE + j]);
    shared_memory[(i+3)*BLOCK_SIZE+j] = min(shared_memory[(i+3)*BLOCK_SIZE+j], shared_memory[(i+3) * BLOCK_SIZE + k]+ shared_memory[k * BLOCK_SIZE + j]);
  }

  // writing data back to global memory
  d_dist[(i+0+r*BLOCK_SIZE) * d_matrix_size + (j+r*BLOCK_SIZE)] = shared_memory[(i+0) * BLOCK_SIZE + j];
  d_dist[(i+1+r*BLOCK_SIZE) * d_matrix_size + (j+r*BLOCK_SIZE)] = shared_memory[(i+1) * BLOCK_SIZE + j];
  d_dist[(i+2+r*BLOCK_SIZE) * d_matrix_size + (j+r*BLOCK_SIZE)] = shared_memory[(i+2) * BLOCK_SIZE + j];
  d_dist[(i+3+r*BLOCK_SIZE) * d_matrix_size + (j+r*BLOCK_SIZE)] = shared_memory[(i+3) * BLOCK_SIZE + j];
}

__global__ void phase2(int* d_dist, int r){
  if (blockIdx.x == r) return;

  __shared__ int pivot[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ int row[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ int col[BLOCK_SIZE][BLOCK_SIZE];

  int j = threadIdx.x;
  int i = threadIdx.y * 4;

  pivot[i+0][j] = d_dist[((i+0) + r*BLOCK_SIZE) * d_matrix_size + (r*BLOCK_SIZE)+j];
  pivot[i+1][j] = d_dist[((i+1) + r*BLOCK_SIZE) * d_matrix_size + (r*BLOCK_SIZE)+j];
  pivot[i+2][j] = d_dist[((i+2) + r*BLOCK_SIZE) * d_matrix_size + (r*BLOCK_SIZE)+j];
  pivot[i+3][j] = d_dist[((i+3) + r*BLOCK_SIZE) * d_matrix_size + (r*BLOCK_SIZE)+j];
  row[i+0][j] = d_dist[((i+0) + r*BLOCK_SIZE) * d_matrix_size + (j + blockIdx.x * BLOCK_SIZE)];
  row[i+1][j] = d_dist[((i+1) + r*BLOCK_SIZE) * d_matrix_size + (j + blockIdx.x * BLOCK_SIZE)];
  row[i+2][j] = d_dist[((i+2) + r*BLOCK_SIZE) * d_matrix_size + (j + blockIdx.x * BLOCK_SIZE)];
  row[i+3][j] = d_dist[((i+3) + r*BLOCK_SIZE) * d_matrix_size + (j + blockIdx.x * BLOCK_SIZE)];
  col[i+0][j] = d_dist[((i+0) + blockIdx.x * BLOCK_SIZE) * d_matrix_size + r * BLOCK_SIZE + j];
  col[i+1][j] = d_dist[((i+1) + blockIdx.x * BLOCK_SIZE) * d_matrix_size + r * BLOCK_SIZE + j];
  col[i+2][j] = d_dist[((i+2) + blockIdx.x * BLOCK_SIZE) * d_matrix_size + r * BLOCK_SIZE + j];
  col[i+3][j] = d_dist[((i+3) + blockIdx.x * BLOCK_SIZE) * d_matrix_size + r * BLOCK_SIZE + j];
  __syncthreads();

  for(int k = 0; k < BLOCK_SIZE; k++){
    row[i+0][j] = min(row[i+0][j], pivot[i+0][k] + row[k][j]);
    row[i+1][j] = min(row[i+1][j], pivot[i+1][k] + row[k][j]);
    row[i+2][j] = min(row[i+2][j], pivot[i+2][k] + row[k][j]);
    row[i+3][j] = min(row[i+3][j], pivot[i+3][k] + row[k][j]);

    col[i+0][j] = min(col[i+0][j], col[i+0][k] + pivot[k][j]);
    col[i+1][j] = min(col[i+1][j], col[i+1][k] + pivot[k][j]);
    col[i+2][j] = min(col[i+2][j], col[i+2][k] + pivot[k][j]);
    col[i+3][j] = min(col[i+3][j], col[i+3][k] + pivot[k][j]);
  }
  __syncthreads();

  d_dist[((i+0) + r*BLOCK_SIZE) * d_matrix_size + (j + blockIdx.x * BLOCK_SIZE)] = row[i+0][j];
  d_dist[((i+1) + r*BLOCK_SIZE) * d_matrix_size + (j + blockIdx.x * BLOCK_SIZE)] = row[i+1][j];
  d_dist[((i+2) + r*BLOCK_SIZE) * d_matrix_size + (j + blockIdx.x * BLOCK_SIZE)] = row[i+2][j];
  d_dist[((i+3) + r*BLOCK_SIZE) * d_matrix_size + (j + blockIdx.x * BLOCK_SIZE)] = row[i+3][j];
  d_dist[((i+0) + blockIdx.x * BLOCK_SIZE) * d_matrix_size + r * BLOCK_SIZE + j] = col[i+0][j];
  d_dist[((i+1) + blockIdx.x * BLOCK_SIZE) * d_matrix_size + r * BLOCK_SIZE + j] = col[i+1][j];
  d_dist[((i+2) + blockIdx.x * BLOCK_SIZE) * d_matrix_size + r * BLOCK_SIZE + j] = col[i+2][j];
  d_dist[((i+3) + blockIdx.x * BLOCK_SIZE) * d_matrix_size + r * BLOCK_SIZE + j] = col[i+3][j];
}

__global__ void phase3(int* d_dist, int r){
  if (blockIdx.x == r || blockIdx.y == r) return;

  __shared__ int row[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ int col[BLOCK_SIZE][BLOCK_SIZE];

  int i = threadIdx.y*4;
  int j = threadIdx.x;

  int i_offset = blockIdx.y * BLOCK_SIZE;
  int j_offset = blockIdx.x * BLOCK_SIZE;
  int block_round = r * BLOCK_SIZE;

  row[i+0][j] = d_dist[(i+0 + i_offset) * d_matrix_size + block_round + j];
  row[i+1][j] = d_dist[(i+1 + i_offset) * d_matrix_size + block_round + j];
  row[i+2][j] = d_dist[(i+2 + i_offset) * d_matrix_size + block_round + j];
  row[i+3][j] = d_dist[(i+3 + i_offset) * d_matrix_size + block_round + j];
  col[i+0][j] = d_dist[(block_round + i+0)*d_matrix_size + (j_offset)+j];
  col[i+1][j] = d_dist[(block_round + i+1)*d_matrix_size + (j_offset)+j];
  col[i+2][j] = d_dist[(block_round + i+2)*d_matrix_size + (j_offset)+j];
  col[i+3][j] = d_dist[(block_round + i+3)*d_matrix_size + (j_offset)+j];

  int i_2_j_0 = d_dist[(i_offset + i+0)*d_matrix_size + (j_offset)+j];
  int i_2_j_1 = d_dist[(i_offset + i+1)*d_matrix_size + (j_offset)+j];
  int i_2_j_2 = d_dist[(i_offset + i+2)*d_matrix_size + (j_offset)+j];
  int i_2_j_3 = d_dist[(i_offset + i+3)*d_matrix_size + (j_offset)+j];

  __syncthreads();

  #pragma unroll 
  for (int k = 0; k < BLOCK_SIZE; k++){
    i_2_j_0 = min(i_2_j_0, row[i+0][k] + col[k][j]);
    i_2_j_1 = min(i_2_j_1, row[i+1][k] + col[k][j]);
    i_2_j_2 = min(i_2_j_2, row[i+2][k] + col[k][j]);
    i_2_j_3 = min(i_2_j_3, row[i+3][k] + col[k][j]);
  }

  d_dist[(i_offset + i+0)*d_matrix_size + (j_offset)+j] = i_2_j_0;
  d_dist[(i_offset + i+1)*d_matrix_size + (j_offset)+j] = i_2_j_1;
  d_dist[(i_offset + i+2)*d_matrix_size + (j_offset)+j] = i_2_j_2;
  d_dist[(i_offset + i+3)*d_matrix_size + (j_offset)+j] = i_2_j_3;

}


void block_FW(int* d_dist) {
  int round = matrix_size/BLOCK_SIZE;
  dim3 phase3_num_blocks(round, round);
  dim3 num_threads(BLOCK_SIZE, BLOCK_SIZE/4);

  // round = 1; // mygodimatomato: for checking
  for (int r = 0; r < round; r++) {
    phase1<<<1, num_threads>>>(d_dist, r);
    phase2<<<round, num_threads>>>(d_dist, r);
    phase3<<<phase3_num_blocks, num_threads>>>(d_dist, r);
  }
}


int main(int argc, char* argv[]) {
  // Read input from input file
  input(argv[1]);

  // Allocate the memory for the matrix in GPU
  int *d_dist;
  hipMalloc((void**)&d_dist, sizeof(int) * matrix_size * matrix_size);
  hipMemcpy(d_dist, adjacency_matrix, sizeof(int) * matrix_size * matrix_size, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_matrix_size), &matrix_size, sizeof(int));
  
  
  // Start executing the block Floyed-Warshall
  block_FW(d_dist);

  // Copy the outcome back to the adjacency_matrix 
  hipMemcpy(adjacency_matrix, d_dist, sizeof(int) * matrix_size * matrix_size, hipMemcpyDeviceToHost);
  
  output(argv[2]);

  // mygodimatomato : for checking
  // int k = 0;
  // for (int i = 0; i < V; i++) {
  //   for (int j = 0; j < V; j++){
  //     if(adjacency_matrix[k] == MY_INF)
  //       printf(" INF ");
  //     else
  //       printf("%4d ", adjacency_matrix[k]);
  //     k++;
  //   } printf("\n");
  // } printf("\n");

  // Write output to output file
  return 0;
}