#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


//======================
#define DEV_NO 0
hipDeviceProp_t prop;





void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j) {
                Dist[i][j] = 0;
            } else {
                Dist[i][j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0]][pair[1]] = pair[2];
    }
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (Dist[i][j] >= INF) Dist[i][j] = INF;
        }
        fwrite(Dist[i], sizeof(int), n, outfile);
    }
    fclose(outfile);
}

int ceil(int a, int b) { return (a + b - 1) / b; }






int main(int argc, char* argv[]) {
    // input(argv[1]);
    int B = 512;

    hipGetDeviceProperties(&prop, DEV_NO);
    printf("maxThreasPerBlock = %d, sharedMemPerBlock = %d", prop.maxThreasPerBlock, prop.sharedMemPerBlock);

    // block_FW(B);
    // output(argv[2]);
    return 0;
}