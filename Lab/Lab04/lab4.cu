
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <zlib.h>
#include <png.h>
#include <hip/hip_fp16.h>

#define Z 2
#define Y 5
#define X 5
#define xBound X / 2
#define yBound Y / 2
#define SCALE 8

int read_png(const char* filename, unsigned char** image, unsigned* height, 
             unsigned* width, unsigned* channels) {

    unsigned char sig[8];
    FILE* infile;
    infile = fopen(filename, "rb");

    fread(sig, 1, 8, infile);
    if (!png_check_sig(sig, 8))
        return 1;   /* bad signature */

    png_structp png_ptr;
    png_infop info_ptr;

    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr)
        return 4;   /* out of memory */
  
    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        return 4;   /* out of memory */
    }

    png_init_io(png_ptr, infile);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, info_ptr);
    int bit_depth, color_type;
    png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

    png_uint_32  i, rowbytes;
    png_bytep  row_pointers[*height];
    png_read_update_info(png_ptr, info_ptr);
    rowbytes = png_get_rowbytes(png_ptr, info_ptr);
    *channels = (int) png_get_channels(png_ptr, info_ptr);

    if ((*image = (unsigned char *) malloc(rowbytes * *height)) == NULL) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        return 3;
    }

    for (i = 0;  i < *height;  ++i)
        row_pointers[i] = *image + i * rowbytes;
    png_read_image(png_ptr, row_pointers);
    png_read_end(png_ptr, NULL);
    return 0;
}

void write_png(const char* filename, png_bytep image, const unsigned height, const unsigned width, 
               const unsigned channels) {
    FILE* fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8,
                 PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
                 PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
    png_write_info(png_ptr, info_ptr);
    png_set_compression_level(png_ptr, 1);

    png_bytep row_ptr[height];
    for (int i = 0; i < height; ++ i) {
        row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
    }
    png_write_image(png_ptr, row_ptr);
    png_write_end(png_ptr, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}

__constant__ char mask[Z][Y][X] = { { { -1, -4, -6, -4, -1 },
                                        { -2, -8, -12, -8, -2 },
                                        { 0, 0, 0, 0, 0 },
                                        { 2, 8, 12, 8, 2 },
                                        { 1, 4, 6, 4, 1 } },
                                      { { -1, -2, 0, 2, 1 },
                                        { -4, -8, 0, 8, 4 },
                                        { -6, -12, 0, 12, 6 },
                                        { -4, -8, 0, 8, 4 },
                                        { -1, -2, 0, 2, 1 } } };

inline __device__ int bound_check(int val, int lower, int upper) {
    if (val >= lower && val < upper)
        return 1;
    else
        return 0;
}
__global__ void sobel(unsigned char *s, unsigned char *t, unsigned height, unsigned width, unsigned channels) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // go from 0 to width - 1
    __half val[Z][3];
    __half zero = __float2half(0.0f);

    if (tid >= width) return;

    int x = tid;
    for (int y = 0; y < height; ++y) {
        for (int i = 0; i < Z; ++i) {
            
            val[i][2] = zero;
            val[i][1] = zero;
            val[i][0] = zero;

            for (int v = -yBound; v <= yBound; ++v) {
                for (int u = -xBound; u <= xBound; ++u) {
                    if (bound_check(x + u, 0, width) && bound_check(y + v, 0, height)) {
                        const __half R = __float2half((float)s[channels * (width * (y + v) + (x + u)) + 2]);
                        const __half G = __float2half((float)s[channels * (width * (y + v) + (x + u)) + 1]);
                        const __half B = __float2half((float)s[channels * (width * (y + v) + (x + u)) + 0]);
                        val[i][2] = __hadd(val[i][2], __hmul(R, __float2half(mask[i][u + xBound][v + yBound])));
                        val[i][1] = __hadd(val[i][1], __hmul(G, __float2half(mask[i][u + xBound][v + yBound])));
                        val[i][0] = __hadd(val[i][0], __hmul(B, __float2half(mask[i][u + xBound][v + yBound])));
                    }
                }
            }
        }
        __half totalR = zero;
        __half totalG = zero;
        __half totalB = zero;
        for (int i = 0; i < Z; ++i) {
            totalR = __hadd(totalR, __hmul(val[i][2], val[i][2]));
            totalG = __hadd(totalG, __hmul(val[i][1], val[i][1]));
            totalB = __hadd(totalB, __hmul(val[i][0], val[i][0]));
        }
        totalR = __hdiv(hsqrt(totalR), __float2half(SCALE));
        totalG = __hdiv(hsqrt(totalG), __float2half(SCALE));
        totalB = __hdiv(hsqrt(totalB), __float2half(SCALE));
        const unsigned char cR = (__half2float(totalR) > 255.f) ? 255 : __half2float(totalR);
        const unsigned char cG = (__half2float(totalG) > 255.f) ? 255 : __half2float(totalG);
        const unsigned char cB = (__half2float(totalB) > 255.f) ? 255 : __half2float(totalB);
        t[channels * (width * y + x) + 2] = cR;
        t[channels * (width * y + x) + 1] = cG;
        t[channels * (width * y + x) + 0] = cB;
    }
}

int main(int argc, char **argv) {
    assert(argc == 3);
    unsigned height, width, channels;
    unsigned char *src = NULL, *dst;
    unsigned char *dsrc, *ddst;

    /* read the image to src, and get height, width, channels */
    if (read_png(argv[1], &src, &height, &width, &channels)) {
        std::cerr << "Error in read png" << std::endl;
        return -1;
    }

    dst = (unsigned char *)malloc(height * width * channels * sizeof(unsigned char));
    hipHostRegister(src, height * width * channels * sizeof(unsigned char), hipHostRegisterDefault);

    // cudaMalloc(...) for device src and device dst
    hipMalloc(&dsrc, height * width * channels * sizeof(unsigned char));
    hipMalloc(&ddst, height * width * channels * sizeof(unsigned char));

    // cudaMemcpy(...) copy source image to device (mask matrix if necessary)
    hipMemcpy(dsrc, src, height * width * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

    // decide to use how many blocks and threads
    const int num_threads = 256;
    // const int num_blocks = height / num_threads + 1;
    const int num_blocks2 = width / num_threads + 1;

    // launch cuda kernel
    // sobel <<<num_blocks2, num_threads>>> (dsrc, ddst, height, width, channels);
    sobel <<< num_blocks2, num_threads >>> (dsrc, ddst, height, width, channels);
    // cudaMemcpy(...) copy result image to host
    hipMemcpy(dst, ddst, height * width * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);

    write_png(argv[2], dst, height, width, channels);
    free(src);
    free(dst);
    hipFree(dsrc);
    hipFree(ddst);
    return 0;
}

